#include "hip/hip_runtime.h"
#include "GridHash.h"
#include "Core/Utility.h"

namespace Physika{

	__constant__ int offset[27][3] = { 0, 0, 0,
		0, 0, 1,
		0, 1, 0,
		1, 0, 0,
		0, 0, -1,
		0, -1, 0,
		-1, 0, 0,
		0, 1, 1,
		0, 1, -1,
		0, -1, 1,
		0, -1, -1,
		1, 0, 1,
		1, 0, -1,
		-1, 0, 1,
		-1, 0, -1,
		1, 1, 0,
		1, -1, 0,
		-1, 1, 0,
		-1, -1, 0,
		1, 1, 1,
		1, 1, -1,
		1, -1, 1,
		-1, 1, 1,
		1, -1, -1,
		-1, 1, -1,
		-1, -1, 1,
		-1, -1, -1
	};

	template<typename TDataType>
	GridHash<TDataType>::GridHash()
	{
	}

	template<typename TDataType>
	GridHash<TDataType>::~GridHash()
	{
	}

	template<typename TDataType>
	void GridHash<TDataType>::setSpace(Real _h, Coord _lo, Coord _hi)
	{
		release();

		int padding = 2;
		ds = _h;
		lo = _lo- padding*ds;

		Coord nSeg = (_hi - _lo) / ds;

		nx = ceil(nSeg[0]) + 1 + 2 * padding;
		ny = ceil(nSeg[1]) + 1 + 2 * padding;
		nz = ceil(nSeg[2]) + 1 + 2 * padding;
		hi = lo + Coord(nx, ny, nz)*ds;

		num = nx*ny*nz;

//		npMax = 128;

		cuSafeCall(hipMalloc((void**)&counter, num * sizeof(int)));
		cuSafeCall(hipMalloc((void**)&index, num * sizeof(int)));
	}

	template<typename TDataType>
	__global__ void K_CalculateParticleNumber(GridHash<TDataType> hash, Array<typename TDataType::Coord> pos)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= pos.size()) return;

		int gId = hash.getIndex(pos[pId]);

		if (gId != INVALID)
			atomicAdd(&(hash.index[gId]), 1);
	}

	template<typename TDataType>
	__global__ void K_ConstructHashTable(GridHash<TDataType> hash, Array<typename TDataType::Coord> pos)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= pos.size()) return;

		int gId = hash.getIndex(pos[pId]);

		if (gId < 0) return;

		int index = atomicAdd(&(hash.counter[gId]), 1);
// 		index = index < hash.npMax - 1 ? index : hash.npMax - 1;
// 		hash.ids[gId * hash.npMax + index] = pId;
		hash.ids[hash.index[gId] + index] = pId;
	}

	template<typename TDataType>
	void GridHash<TDataType>::construct(DeviceArray<Coord>& pos)
	{
		clear();

		dim3 pDims = int(ceil(pos.size() / BLOCK_SIZE + 0.5f));

		K_CalculateParticleNumber << <pDims, BLOCK_SIZE >> > (*this, pos);
		particle_num = thrust::reduce(thrust::device, index, index + num, (int)0, thrust::plus<int>());
		thrust::exclusive_scan(thrust::device, index, index + num, index);

		if (ids != nullptr)
		{
			cuSafeCall(hipFree(ids));
		}
		cuSafeCall(hipMalloc((void**)&ids, particle_num * sizeof(int)));

//		std::cout << "Particle number: " << particle_num << std::endl;

		K_ConstructHashTable << <pDims, BLOCK_SIZE >> > (*this, pos);
		cuSynchronize();
	}

	template<typename TDataType>
	void GridHash<TDataType>::clear()
	{
		cuSafeCall(hipMemset(counter, 0, num * sizeof(int)));
		cuSafeCall(hipMemset(index, 0, num * sizeof(int)));
	}

	template<typename TDataType>
	void GridHash<TDataType>::release()
	{
		if (counter != nullptr)
			cuSafeCall(hipFree(counter));
		
		if (ids != nullptr)
			cuSafeCall(hipFree(ids));

		if (index != nullptr)
			cuSafeCall(hipFree(index));
	}
}