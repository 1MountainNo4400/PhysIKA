#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "HyperelasticForce.h"
#include "Physika_Core/Utilities/Function1Pt.h"
#include "Physika_Framework/Framework/Node.h"
#include "Physika_Core/Algorithm/MatrixFunc.h"
#include "Kernel.h"

namespace Physika
{
	template <typename Real, typename Coord, typename Matrix, typename NPair>
	__global__ void K_ComputeDeformationTensor(
		DeviceArray<Matrix> matArr,
		NeighborList<NPair> restShapes,
		SmoothKernel<Real> kernSmooth,
		Real smoothingLength)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= matArr.size()) return;

		NPair np_i = restShapes.getElement(pId, 0);
		Coord rest_i = np_i.pos;
		int size_i = restShapes.getNeighborSize(pId);

		Real total_weight = 0.0f;
		Matrix mat_i = Matrix(0);
		for (int ne = 0; ne < size_i; ne++)
		{
			NPair np_j = restShapes.getElement(pId, ne);
			Coord rest_j = np_j.pos;
			Real r = (rest_i-rest_j).norm();

			if (r > EPSILON)
			{
				Real weight = kernSmooth.Weight(r, smoothingLength);
				Coord q = (rest_j - rest_i)*sqrt(weight);

				mat_i(0, 0) += q[0] * q[0]; mat_i(0, 1) += q[0] * q[1]; mat_i(0, 2) += q[0] * q[2];
				mat_i(1, 0) += q[1] * q[0]; mat_i(1, 1) += q[1] * q[1]; mat_i(1, 2) += q[1] * q[2];
				mat_i(2, 0) += q[2] * q[0]; mat_i(2, 1) += q[2] * q[1]; mat_i(2, 2) += q[2] * q[2];

				total_weight += weight;
			}
		}

		if (total_weight > EPSILON)
		{
			mat_i *= (1.0f / total_weight);
		}

		Matrix R, U, D;
		polarDecomposition(mat_i, R, U, D);

		Real threshold = 0.0001f*smoothingLength;
		D(0, 0) = D(0, 0) > threshold ? 1.0 / D(0, 0) : 1.0;
		D(1, 1) = D(1, 1) > threshold ? 1.0 / D(1, 1) : 1.0;
		D(2, 2) = D(2, 2) > threshold ? 1.0 / D(2, 2) : 1.0;

		mat_i = R.transpose()*U*D*U.transpose();

		matArr[pId] = mat_i;
	}

/*	template <typename Real, typename Coord, typename Matrix, typename RestShape>
	__global__ void EM_RotateRestShape(
		DeviceArray<Coord> posArr,
		DeviceArray<Matrix> matArr,
		DeviceArray<RestShape> restShapes,
		SmoothKernel<Real> kernSmooth,
		Real smoothingLength)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= posArr.size()) return;

		Coord rest_i = restShapes[pId].pos[restShapes[pId].idx];
		int size_i = restShapes[pId].size;

		//			cout << i << " " << rids[shape_i.ids[shape_i.idx]] << endl;
		Real total_weight = 0.0f;
		Matrix mat_i(0);
		for (int ne = 0; ne < size_i; ne++)
		{
			int j = restShapes[pId].ids[ne];
			float r = restShapes[pId].distance[ne];

			if (r > EPSILON)
			{
				Real weight = kernSmooth.Weight(r, smoothingLength);

				Coord p = posArr[j] - posArr[pId];
				//Vector3f q = (shape_i.pos[ne] - rest_i)*(1.0f/r)*weight;
				float3 q = (restShapes[pId].pos[ne] - rest_i)*weight;

				mat_i(0, 0) += p.x * q.x; mat_i(0, 1) += p.x * q.y; mat_i(0, 2) += p.x * q.z;
				mat_i(1, 0) += p.y * q.x; mat_i(1, 1) += p.y * q.y; mat_i(1, 2) += p.y * q.z;
				mat_i(2, 0) += p.z * q.x; mat_i(2, 1) += p.z * q.y; mat_i(2, 2) += p.z * q.z;
				total_weight += weight;
			}
		}

		if (total_weight > EPSILON)
		{
			mat_i *= (1.0f / total_weight);
			mat_i *= matArr[pId];
		}

		glm::mat3 glmMat3_i;
		glmMat3_i[0][0] = mat_i(0, 0);
		glmMat3_i[0][1] = mat_i(0, 1);
		glmMat3_i[0][2] = mat_i(0, 2);
		glmMat3_i[1][0] = mat_i(1, 0);
		glmMat3_i[1][1] = mat_i(1, 1);
		glmMat3_i[1][2] = mat_i(1, 2);
		glmMat3_i[2][0] = mat_i(2, 0);
		glmMat3_i[2][1] = mat_i(2, 1);
		glmMat3_i[2][2] = mat_i(2, 2);
		glm::mat3 R, U, D;
		PolarDecompositionStable(glmMat3_i, EPSILON, R);

		Matrix matR;
		matR(0, 0) = R[0][0];
		matR(0, 1) = R[0][1];
		matR(0, 2) = R[0][2];
		matR(1, 0) = R[1][0];
		matR(1, 1) = R[1][1];
		matR(1, 2) = R[1][2];
		matR(2, 0) = R[2][0];
		matR(2, 1) = R[2][1];
		matR(2, 2) = R[2][2];

		for (int ne = 0; ne < size_i; ne++)
		{
			int j = restShapes[pId].ids[ne];
			Real r = restShapes[pId].distance[ne];
			if (r > EPSILON)
			{
				Coord v = restShapes[pId].pos[ne] - rest_i;
// 				Coord v3 = Coord(v.x, v.y, v.z);
// 				v3 = matR*v3;
				Coord v3 = matR*v;
				restShapes[pId].pos[ne] = v3 + rest_i;
			}
		}
	}*/

	__device__ Real EM_GetStiffness1(Real r)
	{
		return Real(0.01*(pow(Real(r), Real(3)) - 1));
	}

	template <typename Real, typename Coord, typename Matrix, typename NPair>
	__global__ void K_ComputeElasticForce(
		DeviceArray<Coord> force,
		DeviceArray<Coord> accuPos,
		DeviceArray<Real> accuLamdas,
		DeviceArray<Real> bulkCoefs,
		DeviceArray<Coord> posArr,
		DeviceArray<Matrix> matArr,
		NeighborList<NPair> restShapes,
		SmoothKernel<Real> kernSmooth,
		Real smoothingLength)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= posArr.size()) return;

		NPair np_i = restShapes.getElement(pId, 0);
		Coord rest_i = np_i.pos;
		int size_i = restShapes.getNeighborSize(pId);
		Coord pos_i = posArr[pId];

		//			cout << i << " " << rids[shape_i.ids[shape_i.idx]] << endl;
		Real total_weight = 0.0f;
		Matrix deform_i = Matrix(0.0f);
		for (int ne = 0; ne < size_i; ne++)
		{
			NPair np_j = restShapes.getElement(pId, ne);
			Coord rest_j = np_j.pos;
			int j = np_j.j;

			Real r = (rest_j - rest_i).norm();

			if (r > EPSILON)
			{
				Real weight = kernSmooth.Weight(r, smoothingLength);

				Coord p = (posArr[j] - posArr[pId]);
				Coord q = (rest_j - rest_i)*weight;

				deform_i(0, 0) += p[0] * q[0]; deform_i(0, 1) += p[0] * q[1]; deform_i(0, 2) += p[0] * q[2];
				deform_i(1, 0) += p[1] * q[0]; deform_i(1, 1) += p[1] * q[1]; deform_i(1, 2) += p[1] * q[2];
				deform_i(2, 0) += p[2] * q[0]; deform_i(2, 1) += p[2] * q[1]; deform_i(2, 2) += p[2] * q[2];
				total_weight += weight;
			}
		}

		if (total_weight > EPSILON)
		{
			deform_i *= (1.0f / total_weight);
			//deform_i *= matArr[pId];
			deform_i = deform_i * matArr[pId];
		}
		else
		{
			total_weight = 1.0f;
		}

		if ((deform_i.determinant()) < 0.01f)
		{
			deform_i = Matrix::identityMatrix();
		}

		Matrix mat_i = deform_i;

		for (int ne = 0; ne < size_i; ne++)
		{
			NPair np_j = restShapes.getElement(pId, ne);
			Coord rest_j = np_j.pos;
			int j = np_j.j;
			Real r = (rest_j - rest_i).norm();
			Coord pos_j = posArr[j];
			Real l = (pos_j - pos_i).norm();
			
			if (r > EPSILON)
			{
				Real weight = kernSmooth.Weight(r, smoothingLength);

				Coord q = (rest_j - rest_i)*(1.0f / r);
				Coord p = mat_i*q;
				Coord p_ij = (pos_j-pos_i);

				p.normalize();
				p_ij.normalize();

				p = 0.7*p_ij -0.5*p;

				Real bulk_ij = EM_GetStiffness1(l / r)*weight;

				Coord df_i = bulk_ij*p;
				Coord df_j = -bulk_ij*p;

				atomicAdd(&force[pId][0], df_i[0]);
				atomicAdd(&force[pId][1], df_i[1]);
				atomicAdd(&force[pId][2], df_i[2]);
				atomicAdd(&force[j][0], df_j[0]);
				atomicAdd(&force[j][1], df_j[1]);
				atomicAdd(&force[j][2], df_j[2]);

			}
		}
	}

	template <typename Real, typename Coord>
	__global__ void K_UpdatePosition1(
		DeviceArray<Coord> posArr,
		DeviceArray<Coord> tmpPos,
		DeviceArray<Coord> accuPos,
		DeviceArray<Real> accuLambda)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= posArr.size()) return;

		posArr[pId] = (tmpPos[pId] + accuPos[pId]) / (1.0f + accuLambda[pId]);

// 		if (pId % 200 == 0)
// 		{
// 			printf("%f %f %f \n", accuPos[pId][0], accuPos[pId][1], accuPos[pId][2]);
// 		}
	}

	template <typename Real, typename Coord>
	__global__ void K_UpdateVelocity1(
		DeviceArray<Coord> velArr,
		DeviceArray<Coord> prePos,
		DeviceArray<Coord> curPos,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= velArr.size()) return;

		velArr[pId] *= 0.8;
	}

	template<typename TDataType>
	HyperelasticForce<TDataType>::HyperelasticForce()
		: ForceModule()
		, m_posID(MechanicalState::position())
		, m_velID(MechanicalState::velocity())
		, m_forceID(MechanicalState::force())
		, m_neighborhoodID(MechanicalState::particle_neighbors())
		, m_initPosID(MechanicalState::init_position())
		, m_posPreID(MechanicalState::pre_position())
		, m_refMatrix(NULL)
		, m_tmpPos(NULL)
		, m_lambdas(NULL)
		, m_accPos(NULL)
		, m_bulkCoef(NULL)
		, m_needUpdate(true)
		, m_horizon(Real(0.0125))
	{
	}

	template<typename TDataType>
	bool HyperelasticForce<TDataType>::applyForce()
	{
		Real dt = getParent()->getDt();

		auto mstate = getParent()->getMechanicalState();
		if (!mstate)
		{
			std::cout << "Cannot find a parent node for SummationDensity!" << std::endl;
		}

		auto posFd = mstate->getField<DeviceArrayField<Coord>>(m_posID);
		auto velFd = mstate->getField<DeviceArrayField<Coord>>(m_velID);
		auto forceFd = mstate->getField<DeviceArrayField<Coord>>(m_forceID);
		auto posPreFd = mstate->getField<DeviceArrayField<Coord>>(m_posPreID);
		auto neighborFd = mstate->getField<NeighborField<int>>(m_neighborhoodID);


		uint pDims = cudaGridSize(posFd->getReference()->size(), BLOCK_SIZE);

		int num = posFd->getReference()->size();
		if (NULL == m_refMatrix)
			m_refMatrix = DeviceArrayField<Matrix>::create(num);
		if (NULL == m_tmpPos)
			m_tmpPos = DeviceArrayField<Coord>::create(num);
		if (NULL == m_lambdas)
			m_lambdas = DeviceArrayField<Real>::create(num);
		if (NULL == m_accPos)
			m_accPos = DeviceArrayField<Coord>::create(num);
		if (NULL == m_bulkCoef)
			m_bulkCoef = DeviceArrayField<Real>::create(num);


		auto matArr = m_refMatrix->getReference();
		auto lambda = m_lambdas->getReference();
		auto bulks = m_bulkCoef->getReference();
		auto accPos = m_accPos->getReference();
		auto tmpPos = m_tmpPos->getReference();

		if (isUpdateRequired())
		{
			construct(neighborFd->getValue(), posFd->getValue());
		}

		Function1Pt::copy(*tmpPos, posFd->getValue());
		K_ComputeDeformationTensor <Real, Coord, Matrix, NPair> << <pDims, BLOCK_SIZE >> > (*matArr, m_refPos, SmoothKernel<Real>(), m_horizon);

		int total_itoration = 5;
		int itor = 0;
		accPos->reset();
		lambda->reset();
		K_ComputeElasticForce << <pDims, BLOCK_SIZE >> > (
			forceFd->getValue(),
			*accPos,
			*lambda,
			*bulks,
			posFd->getValue(),
			*matArr,
			m_refPos,
			SmoothKernel<Real>(),
			m_horizon);

		K_UpdateVelocity1 << <pDims, BLOCK_SIZE >> > (velFd->getValue(), posPreFd->getValue(), posFd->getValue(), dt);

		return true;
	}

	template <typename Coord, typename NPair>
	__global__ void K_UpdateRestShape1(
		NeighborList<NPair> shape,
		NeighborList<int> nbr,
		DeviceArray<Coord> pos)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= pos.size()) return;

		NPair np;
		int nbSize = nbr.getNeighborSize(pId);
		
		for (int ne = 0; ne < nbSize; ne++)
		{
			int j = nbr.getElement(pId, ne);
			np.j = j;
			np.pos = pos[j];
 			if (pId != j)
 			{
 				shape.setElement(pId, ne, np);
			}
			else
			{
				if (ne == 0)
				{
					shape.setElement(pId, ne, np);
				}
				else
				{
					auto ele = shape.getElement(pId, 0);
					shape.setElement(pId, 0, np);
					shape.setElement(pId, ne, ele);
				}
			}
		}
	}

	template<typename TDataType>
	void HyperelasticForce<TDataType>::construct(NeighborList<int>& nbr, DeviceArray<Coord>& pos)
	{
		m_refPos.resize(nbr.size());
		if (nbr.isLimited())
		{
			m_refPos.setNeighborLimit(nbr.getNeighborLimit());
		}

		Function1Pt::copy(m_refPos.getIndex(), nbr.getIndex());

		uint pDims = cudaGridSize(pos.size(), BLOCK_SIZE);

		K_UpdateRestShape1 << <pDims, BLOCK_SIZE >> > (m_refPos, nbr, pos);

		m_needUpdate = false;
	}
}