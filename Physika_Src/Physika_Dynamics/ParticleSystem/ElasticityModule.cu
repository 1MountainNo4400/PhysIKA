#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ElasticityModule.h"
#include "Physika_Core/Utilities/Function1Pt.h"
#include "Physika_Framework/Framework/Node.h"
#include "Physika_Core/Algorithm/MatrixFunc.h"
#include "Kernel.h"

namespace Physika
{
	template <typename Real, typename Coord, typename Matrix, typename NPair>
	__global__ void EM_PrecomputeShape(
		DeviceArray<Matrix> matArr,
		NeighborList<NPair> restShapes,
		SmoothKernel<Real> kernSmooth,
		Real smoothingLength)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= matArr.size()) return;

		NPair np_i = restShapes.getElement(pId, 0);
		Coord rest_i = np_i.pos;
		int size_i = restShapes.getNeighborSize(pId);

		Real total_weight = 0.0f;
		Matrix mat_i = Matrix(0);
		for (int ne = 0; ne < size_i; ne++)
		{
			NPair np_j = restShapes.getElement(pId, ne);
			Coord rest_j = np_j.pos;
			Real r = (rest_i-rest_j).norm();

			if (r > EPSILON)
			{
				Real weight = kernSmooth.Weight(r, smoothingLength);
				Coord q = (rest_j - rest_i)*sqrt(weight);

				mat_i(0, 0) += q[0] * q[0]; mat_i(0, 1) += q[0] * q[1]; mat_i(0, 2) += q[0] * q[2];
				mat_i(1, 0) += q[1] * q[0]; mat_i(1, 1) += q[1] * q[1]; mat_i(1, 2) += q[1] * q[2];
				mat_i(2, 0) += q[2] * q[0]; mat_i(2, 1) += q[2] * q[1]; mat_i(2, 2) += q[2] * q[2];

				total_weight += weight;
			}
		}

		if (total_weight > EPSILON)
		{
			mat_i *= (1.0f / total_weight);
		}

		Matrix R, U, D;
		polarDecomposition(mat_i, R, U, D);

		Real threshold = 0.0001f*smoothingLength;
		D(0, 0) = D(0, 0) > threshold ? 1.0 / D(0, 0) : 1.0;
		D(1, 1) = D(1, 1) > threshold ? 1.0 / D(1, 1) : 1.0;
		D(2, 2) = D(2, 2) > threshold ? 1.0 / D(2, 2) : 1.0;

		mat_i = R.transpose()*U*D*U.transpose();

		matArr[pId] = mat_i;
	}

/*	template <typename Real, typename Coord, typename Matrix, typename RestShape>
	__global__ void EM_RotateRestShape(
		DeviceArray<Coord> posArr,
		DeviceArray<Matrix> matArr,
		DeviceArray<RestShape> restShapes,
		SmoothKernel<Real> kernSmooth,
		Real smoothingLength)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= posArr.size()) return;

		Coord rest_i = restShapes[pId].pos[restShapes[pId].idx];
		int size_i = restShapes[pId].size;

		//			cout << i << " " << rids[shape_i.ids[shape_i.idx]] << endl;
		Real total_weight = 0.0f;
		Matrix mat_i(0);
		for (int ne = 0; ne < size_i; ne++)
		{
			int j = restShapes[pId].ids[ne];
			float r = restShapes[pId].distance[ne];

			if (r > EPSILON)
			{
				Real weight = kernSmooth.Weight(r, smoothingLength);

				Coord p = posArr[j] - posArr[pId];
				//Vector3f q = (shape_i.pos[ne] - rest_i)*(1.0f/r)*weight;
				float3 q = (restShapes[pId].pos[ne] - rest_i)*weight;

				mat_i(0, 0) += p.x * q.x; mat_i(0, 1) += p.x * q.y; mat_i(0, 2) += p.x * q.z;
				mat_i(1, 0) += p.y * q.x; mat_i(1, 1) += p.y * q.y; mat_i(1, 2) += p.y * q.z;
				mat_i(2, 0) += p.z * q.x; mat_i(2, 1) += p.z * q.y; mat_i(2, 2) += p.z * q.z;
				total_weight += weight;
			}
		}

		if (total_weight > EPSILON)
		{
			mat_i *= (1.0f / total_weight);
			mat_i *= matArr[pId];
		}

		glm::mat3 glmMat3_i;
		glmMat3_i[0][0] = mat_i(0, 0);
		glmMat3_i[0][1] = mat_i(0, 1);
		glmMat3_i[0][2] = mat_i(0, 2);
		glmMat3_i[1][0] = mat_i(1, 0);
		glmMat3_i[1][1] = mat_i(1, 1);
		glmMat3_i[1][2] = mat_i(1, 2);
		glmMat3_i[2][0] = mat_i(2, 0);
		glmMat3_i[2][1] = mat_i(2, 1);
		glmMat3_i[2][2] = mat_i(2, 2);
		glm::mat3 R, U, D;
		PolarDecompositionStable(glmMat3_i, EPSILON, R);

		Matrix matR;
		matR(0, 0) = R[0][0];
		matR(0, 1) = R[0][1];
		matR(0, 2) = R[0][2];
		matR(1, 0) = R[1][0];
		matR(1, 1) = R[1][1];
		matR(1, 2) = R[1][2];
		matR(2, 0) = R[2][0];
		matR(2, 1) = R[2][1];
		matR(2, 2) = R[2][2];

		for (int ne = 0; ne < size_i; ne++)
		{
			int j = restShapes[pId].ids[ne];
			Real r = restShapes[pId].distance[ne];
			if (r > EPSILON)
			{
				Coord v = restShapes[pId].pos[ne] - rest_i;
// 				Coord v3 = Coord(v.x, v.y, v.z);
// 				v3 = matR*v3;
				Coord v3 = matR*v;
				restShapes[pId].pos[ne] = v3 + rest_i;
			}
		}
	}*/

	__device__ float EM_GetStiffness(int r)
	{
		return 10.0f;
	}

	template <typename Real, typename Coord, typename Matrix, typename NPair>
	__global__ void EM_EnforceElasticity(
		DeviceArray<Coord> accuPos,
		DeviceArray<Real> accuLamdas,
		DeviceArray<Real> bulkCoefs,
		DeviceArray<Coord> posArr,
		DeviceArray<Matrix> matArr,
		NeighborList<NPair> restShapes,
		SmoothKernel<Real> kernSmooth,
		Real smoothingLength)
	{

		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= posArr.size()) return;

		NPair np_i = restShapes.getElement(pId, 0);
		Coord rest_i = np_i.pos;
		int size_i = restShapes.getNeighborSize(pId);

		//			cout << i << " " << rids[shape_i.ids[shape_i.idx]] << endl;
		Real total_weight = 0.0f;
		Matrix deform_i = Matrix(0.0f);
		for (int ne = 0; ne < size_i; ne++)
		{
			NPair np_j = restShapes.getElement(pId, ne);
			Coord rest_j = np_j.pos;
			int j = np_j.j;

			Real r = (rest_j - rest_i).norm();

			if (r > EPSILON)
			{
				Real weight = kernSmooth.Weight(r, smoothingLength);

				Coord p = (posArr[j] - posArr[pId]);
				Coord q = (rest_j - rest_i)*weight;

				deform_i(0, 0) += p[0] * q[0]; deform_i(0, 1) += p[0] * q[1]; deform_i(0, 2) += p[0] * q[2];
				deform_i(1, 0) += p[1] * q[0]; deform_i(1, 1) += p[1] * q[1]; deform_i(1, 2) += p[1] * q[2];
				deform_i(2, 0) += p[2] * q[0]; deform_i(2, 1) += p[2] * q[1]; deform_i(2, 2) += p[2] * q[2];
				total_weight += weight;
			}
		}

		if (total_weight > EPSILON)
		{
			deform_i *= (1.0f / total_weight);
			//deform_i *= matArr[pId];
			deform_i = deform_i * matArr[pId];
		}
		else
		{
			total_weight = 1.0f;
		}

		if ((deform_i.determinant()) < 0.01f)
		{
			deform_i = Matrix::identityMatrix();
		}

		Matrix mat_i = deform_i;

		for (int ne = 0; ne < size_i; ne++)
		{
			NPair np_j = restShapes.getElement(pId, ne);
			Coord rest_j = np_j.pos;
			int j = np_j.j;
			Real r = (rest_j - rest_i).norm();

			if (r > EPSILON)
			{
				Real weight = kernSmooth.Weight(r, smoothingLength);

				Coord q = (rest_i - rest_j)*(1.0f / r);
				//Coord p = Vec2Float(Float2Vec(q)*mat_i);
				Coord p = mat_i*q;
				//p = normalize(p);

				p.normalize();

				Coord dir_ij = 1.0f*r*p;
				// 					Vector3f q = (rest_i - shape_i.pos[ne]);//*(1.0f/r);
				// 					Vector3f p = mat_i*q;
				// 					//p.Normalize();
				// 					Vector3f dir_ij = 1.0f*p;
				Coord new_pos_i = dir_ij + posArr[j];
				Coord new_pos_j = -dir_ij + posArr[pId];
				Coord dir_i = 1.0f*(new_pos_i - posArr[pId]);//*(samplingDistance/r)*(samplingDistance/r);
				Coord dir_j = 1.0f*(new_pos_j - posArr[j]);//*(samplingDistance/r)*(samplingDistance/r);

				Real l_i = dir_i.norm();

				Real ratio = weight / total_weight;
				Real cc = (smoothingLength / r);
				if (r < 0.8*smoothingLength)
				{
					cc = 1.0 / 0.8f;
				}

				Real bulk_ij = 1.0f*EM_GetStiffness(l_i / r)*ratio*cc*cc;
				Coord vec_ij = bulk_ij*dir_i;

				atomicAdd(&accuLamdas[pId], bulk_ij);
				atomicAdd(&accuLamdas[j], bulk_ij);

				Coord dP_i = bulk_ij*new_pos_i;
				Coord dP_j = bulk_ij*new_pos_j;

				atomicAdd(&accuPos[pId][0], dP_i[0]);
				atomicAdd(&accuPos[pId][1], dP_i[1]);
				atomicAdd(&accuPos[pId][2], dP_i[2]);
				atomicAdd(&accuPos[j][0], dP_j[0]);
				atomicAdd(&accuPos[j][1], dP_j[1]);
				atomicAdd(&accuPos[j][2], dP_j[2]);

			}
		}
	}

	template <typename Real, typename Coord>
	__global__ void K_UpdatePosition(
		DeviceArray<Coord> posArr,
		DeviceArray<Coord> tmpPos,
		DeviceArray<Coord> accuPos,
		DeviceArray<Real> accuLambda)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= posArr.size()) return;

		posArr[pId] = (tmpPos[pId] + accuPos[pId]) / (1.0f + accuLambda[pId]);

// 		if (pId % 200 == 0)
// 		{
// 			printf("%f %f %f \n", accuPos[pId][0], accuPos[pId][1], accuPos[pId][2]);
// 		}
	}

	template <typename Real, typename Coord>
	__global__ void K_UpdateVelocity(
		DeviceArray<Coord> velArr,
		DeviceArray<Coord> prePos,
		DeviceArray<Coord> curPos,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= velArr.size()) return;

		velArr[pId] = (curPos[pId] - prePos[pId]) / dt;
	}

	template<typename TDataType>
	ElasticityModule<TDataType>::ElasticityModule()
		: ConstraintModule()
		, m_neighborhoodID(MechanicalState::particle_neighbors())
		, m_initPosID(MechanicalState::init_position())
		, m_posPreID(MechanicalState::pre_position())
		, m_refMatrix(NULL)
		, m_tmpPos(NULL)
		, m_lambdas(NULL)
		, m_accPos(NULL)
		, m_bulkCoef(NULL)
		, m_needUpdate(true)
		, m_horizon(Real(0.0125))
	{
	}

	template<typename TDataType>
	bool ElasticityModule<TDataType>::constrain()
	{
		Real dt = getParent()->getDt();

		auto mstate = getParent()->getMechanicalState();
		if (!mstate)
		{
			std::cout << "Cannot find a parent node for SummationDensity!" << std::endl;
		}

		auto posFd = mstate->getField<DeviceArrayField<Coord>>(m_posID);
		auto velFd = mstate->getField<DeviceArrayField<Coord>>(m_velID);
		auto posPreFd = mstate->getField<DeviceArrayField<Coord>>(m_posPreID);
		auto neighborFd = mstate->getField<NeighborField<int>>(m_neighborhoodID);


		uint pDims = cudaGridSize(posFd->size(), BLOCK_SIZE);

		int num = posFd->size();
		if (NULL == m_refMatrix)
			m_refMatrix = DeviceArrayField<Matrix>::create(num);
		if (NULL == m_tmpPos)
			m_tmpPos = DeviceArrayField<Coord>::create(num);
		if (NULL == m_lambdas)
			m_lambdas = DeviceArrayField<Real>::create(num);
		if (NULL == m_accPos)
			m_accPos = DeviceArrayField<Coord>::create(num);
		if (NULL == m_bulkCoef)
			m_bulkCoef = DeviceArrayField<Real>::create(num);


		DeviceArray<Matrix>* matArr = m_refMatrix->getDataPtr();
		DeviceArray<Real>* lambda = m_lambdas->getDataPtr();
		DeviceArray<Real>* bulks = m_bulkCoef->getDataPtr();
		DeviceArray<Coord>* accPos = m_accPos->getDataPtr();
		DeviceArray<Coord>* tmpPos = m_tmpPos->getDataPtr();

		if (isUpdateRequired())
		{
			construct(neighborFd->getValue(), posFd->getValue());
		}

		Function1Pt::copy(*tmpPos, posFd->getValue());
		EM_PrecomputeShape <Real, Coord, Matrix, NPair> << <pDims, BLOCK_SIZE >> > (*matArr, m_refPos, SmoothKernel<Real>(), m_horizon);

		int total_itoration = 5;
		int itor = 0;
		while (itor < total_itoration)
		{
			accPos->reset();
			lambda->reset();
			EM_EnforceElasticity << <pDims, BLOCK_SIZE >> > (
				*accPos,
				*lambda,
				*bulks,
				posFd->getValue(),
				*matArr,
				m_refPos,
				SmoothKernel<Real>(),
				m_horizon);
			K_UpdatePosition << <pDims, BLOCK_SIZE >> > (posFd->getValue(), *tmpPos, *accPos, *lambda);
			itor++;
		}

//		EM_RotateRestShape << <pDims, BLOCK_SIZE >> > (*posArr, *matArr, *restShapeArr);

		K_UpdateVelocity << <pDims, BLOCK_SIZE >> > (velFd->getValue(), posPreFd->getValue(), posFd->getValue(), dt);

		return true;
	}

	template <typename Coord, typename NPair>
	__global__ void K_UpdateRestShape(
		NeighborList<NPair> shape,
		NeighborList<int> nbr,
		DeviceArray<Coord> pos)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= pos.size()) return;

		NPair np;
		int nbSize = nbr.getNeighborSize(pId);
		
		for (int ne = 0; ne < nbSize; ne++)
		{
			int j = nbr.getElement(pId, ne);
			np.j = j;
			np.pos = pos[j];
 			if (pId != j)
 			{
 				shape.setElement(pId, ne, np);
			}
			else
			{
				if (ne == 0)
				{
					shape.setElement(pId, ne, np);
				}
				else
				{
					auto ele = shape.getElement(pId, 0);
					shape.setElement(pId, 0, np);
					shape.setElement(pId, ne, ele);
				}
			}
		}
	}

	template<typename TDataType>
	void ElasticityModule<TDataType>::construct(NeighborList<int>& nbr, DeviceArray<Coord>& pos)
	{
		m_refPos.resize(nbr.size());
		if (nbr.isLimited())
		{
			m_refPos.setNeighborLimit(nbr.getNeighborLimit());
		}

		Function1Pt::copy(m_refPos.getIndex(), nbr.getIndex());

		uint pDims = cudaGridSize(pos.size(), BLOCK_SIZE);

		K_UpdateRestShape<< <pDims, BLOCK_SIZE >> > (m_refPos, nbr, pos);

		m_needUpdate = false;
	}
}