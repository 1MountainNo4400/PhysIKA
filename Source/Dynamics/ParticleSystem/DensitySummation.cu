#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "DensitySummation.h"
#include "Framework/Framework/MechanicalState.h"
#include "Framework/Framework/Node.h"
#include "Core/Utility.h"
#include "Kernel.h"

namespace PhysIKA
{
	IMPLEMENT_CLASS_1(DensitySummation, TDataType)

	template<typename Real, typename Coord>
	__global__ void K_ComputeDensity(
		DeviceArray<Real> rhoArr,
		DeviceArray<Coord> posArr,
		NeighborList<int> neighbors,
		Real smoothingLength,
		Real mass
	)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= posArr.size()) return;

		SpikyKernel<Real> kern;
		Real r;
		Real rho_i = Real(0);
		Coord pos_i = posArr[pId];
		int nbSize = neighbors.getNeighborSize(pId);
		for (int ne = 0; ne < nbSize; ne++)
		{
			int j = neighbors.getElement(pId, ne);
			r = (pos_i - posArr[j]).norm();
			rho_i += mass*kern.Weight(r, smoothingLength);
		}
		rhoArr[pId] = rho_i;
	}

	template<typename TDataType>
	DensitySummation<TDataType>::DensitySummation()
		: ComputeModule()
		, m_factor(Real(0.000044))
	{
		m_mass.setValue(Real(1));
		m_restDensity.setValue(Real(1000));
		m_smoothingLength.setValue(Real(0.011));

		attachField(&m_mass, "mass", "particle mass", false);
		attachField(&m_restDensity, "rest_density", "Reference density", false);
		attachField(&m_smoothingLength, "smoothing_length", "The smoothing length in SPH!", false);

		attachField(&m_position, "position", "Storing the particle positions!", false);
		attachField(&m_density, "density", "Storing the particle densities!", false);
		attachField(&m_neighborhood, "neighborhood", "Storing neighboring particles' ids!", false);
	}

	template<typename TDataType>
	void DensitySummation<TDataType>::compute()
	{
		compute(
			m_density.getValue(),
			m_position.getValue(),
			m_neighborhood.getValue(),
			m_smoothingLength.getValue(),
			m_mass.getValue());
	}


	template<typename TDataType>
	void DensitySummation<TDataType>::compute(DeviceArray<Real>& rho)
	{
		compute(
			rho,
			m_position.getValue(),
			m_neighborhood.getValue(),
			m_smoothingLength.getValue(),
			m_mass.getValue());
	}

	template<typename TDataType>
	void DensitySummation<TDataType>::compute(
		DeviceArray<Real>& rho, 
		DeviceArray<Coord>& pos,
		NeighborList<int>& neighbors, 
		Real smoothingLength,
		Real mass)
	{
		cuint pDims = cudaGridSize(rho.size(), BLOCK_SIZE);
		K_ComputeDensity <Real, Coord> << <pDims, BLOCK_SIZE >> > (rho, pos, neighbors, smoothingLength, m_factor*mass);
	}

	template<typename TDataType>
	bool DensitySummation<TDataType>::initializeImpl()
	{
		if (!m_position.isEmpty() && m_density.isEmpty())
		{
			m_density.setElementCount(m_position.getElementCount());
		}

		if (!isAllFieldsReady())
		{
			std::cout << "Exception: " << std::string("DensitySummation's fields are not fully initialized!") << "\n";
			return false;
		}
		
		compute(
			m_density.getValue(),
			m_position.getValue(),
			m_neighborhood.getValue(),
			m_smoothingLength.getValue(),
			m_mass.getValue());
		

		Real sampling_distance = 0.005;
		int sum = m_smoothingLength.getValue() / sampling_distance;
		sum += 2;

		SpikyKernel<Real> kern;
		Real rho_i(0);
		for(int i = -sum; i <= sum; i ++)
			for (int j = -sum; j <= sum; j++)
				for (int k = -sum; k <= sum; k++)
				{
					Real x = i * sampling_distance;
					Real y = j * sampling_distance;
					Real z = k * sampling_distance;
					Real r = sqrt(x * x + y * y + z * z);
					rho_i += m_mass.getValue() * kern.Weight(r, m_smoothingLength.getValue());
				}	

	//	printf("RHO:          %.10lf\n", rho_i);
		auto rho = m_density.getReference();

		Reduction<Real>* pReduce = Reduction<Real>::Create(rho->size());

		//Real maxRho = pReduce->maximum(rho->getDataPtr(), rho->size());
		
		Real maxRho = rho_i;
		//printf("RHO2:          %.10lf\n", maxRho);
		m_factor = m_restDensity.getValue() / maxRho;
		
//		delete pReduce;

		return true;
	}
}