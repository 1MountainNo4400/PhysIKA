#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ParticleIntegrator.h"
#include "Framework/Framework/FieldArray.h"
#include "Framework/Framework/FieldVar.h"
#include "Framework/Framework/Node.h"
#include "Core/Utility.h"
#include "Framework/Framework/SceneGraph.h"

namespace PhysIKA
{
	IMPLEMENT_CLASS_1(ParticleIntegrator, TDataType)

	template<typename TDataType>
	ParticleIntegrator<TDataType>::ParticleIntegrator()
		: NumericalIntegrator()
	{
	}

	template<typename TDataType>
	void ParticleIntegrator<TDataType>::begin()
	{
		Function1Pt::copy(m_prePosition, this->inPosition()->getValue());
		Function1Pt::copy(m_preVelocity, this->inVelocity()->getValue());
		
		this->inForceDensity()->getReference()->reset();
	}

	template<typename TDataType>
	void ParticleIntegrator<TDataType>::end()
	{

	}

	template<typename TDataType>
	bool ParticleIntegrator<TDataType>::initializeImpl()
	{
		if (!isAllFieldsReady())
		{
			std::cout << "Exception: " << std::string("DensitySummation's fields are not fully initialized!") << "\n";
			return false;
		}

		int num = this->inPosition()->getElementCount();

		m_prePosition.resize(num);
		m_preVelocity.resize(num);

		return true;
	}

	template<typename Real, typename Coord>
	__global__ void K_UpdateVelocity(
		DeviceArray<Coord> vel,
		DeviceArray<Coord> forceDensity,
		Coord gravity,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= forceDensity.size()) return;

		vel[pId] += dt * (forceDensity[pId] + gravity);
	}


	template<typename Real, typename Coord>
	__global__ void K_UpdateVelocity(
		DeviceArray<Coord> vel,
		DeviceArray<Coord> force,
		DeviceArray<Real> mass,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= force.size()) return;

		vel[pId] += dt * force[pId] / mass[pId];
	}

	template<typename TDataType>
	bool ParticleIntegrator<TDataType>::updateVelocity()
	{
		Real dt = getParent()->getDt();
		Coord gravity = SceneGraph::getInstance().getGravity();
		cuint pDims = cudaGridSize(this->inPosition()->getReference()->size(), BLOCK_SIZE);

		K_UpdateVelocity << <pDims, BLOCK_SIZE >> > (
			this->inVelocity()->getValue(), 
			this->inForceDensity()->getValue(),
			gravity,
			dt);

		return true;
	}

	template<typename Real, typename Coord>
	__global__ void K_UpdatePosition(
		DeviceArray<Coord> pos,
		DeviceArray<Coord> vel,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= pos.size()) return;

		pos[pId] += dt * vel[pId];
	}

	template<typename TDataType>
	bool ParticleIntegrator<TDataType>::updatePosition()
	{
		Real dt = getParent()->getDt();
		cuint pDims = cudaGridSize(this->inPosition()->getReference()->size(), BLOCK_SIZE);

		K_UpdatePosition << <pDims, BLOCK_SIZE >> > (
			this->inPosition()->getValue(), 
			this->inVelocity()->getValue(), 
			dt);

		return true;
	}

	template<typename TDataType>
	bool ParticleIntegrator<TDataType>::integrate()
	{
		updateVelocity();
		updatePosition();

		return true;
	}
}