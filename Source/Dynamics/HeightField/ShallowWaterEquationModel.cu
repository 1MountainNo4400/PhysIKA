#include "hip/hip_runtime.h"
#include "ShallowWaterEquationModel.h"
#include "Framework/Topology/PointSet.h"
#include "Framework/Framework/Node.h"
#include "Framework/Framework/MechanicalState.h"
#include "Framework/Mapping/PointSetToPointSet.h"
#include "Framework/Topology/FieldNeighbor.h"
#include "Framework/Topology/NeighborQuery.h"
#include "Dynamics/ParticleSystem/Helmholtz.h"
#include "Dynamics/ParticleSystem/Attribute.h"
#include "Core/Utility.h"
#include <hip/hip_runtime.h>
#include "hipblas.h"
namespace PhysIKA
{
	IMPLEMENT_CLASS_1(ShallowWaterEquationModel, TDataType)

	template<typename TDataType>
	ShallowWaterEquationModel<TDataType>::ShallowWaterEquationModel()
		: NumericalModel()
		, m_pNum(0)
	{
		attachField(&m_position, "position", "Storing the particle positions!", false);
		attachField(&m_velocity, "velocity", "Storing the particle velocities!", false);
		attachField(&grid_vel_x, "grid_vel_x", "Storing the grid velocities x!", false);
		attachField(&grid_vel_z, "grid_vel_z", "Storing the grid velocities z!", false);

		//attachField(&m_force, "force", "Storing the particle force densities!", false);

		attachField(&solid, "solid", "Storing the solid grid!", false);
		attachField(&normal, "solidnormal", "Storing the solid normal!", false);
		attachField(&isBound, "isBound", "Storing the solid isBound!", false);
		attachField(&h, "h", "Storing the water height!", false);
	}

	template<typename Real, typename Coord>
	__global__ void Init(
		DeviceArray<Coord> pos,
		DeviceArray<Coord> solid,
		DeviceArray<Real> h,
		DeviceArray<int> isBound,
		DeviceArray<Coord> m_velocity
		)
	{
		int i = threadIdx.x + (blockIdx.x * blockDim.x);
		if (i >= pos.size()) return;
	
		h[i] = pos[i][1] - solid[i][1];
		m_velocity[i] = Coord(0, 0, 0);
	}

	template<typename Real, typename Coord>
	__global__ void Init_gridVel(
		DeviceArray<Real> grid_vel_x,
		DeviceArray<Real> grid_vel_z
	)
	{
		int i = threadIdx.x + (blockIdx.x * blockDim.x);
		if (i < grid_vel_x.size()) grid_vel_x[i] = 0;
		if (i < grid_vel_z.size()) grid_vel_z[i] = 0;

	}
	template<typename TDataType>
	ShallowWaterEquationModel<TDataType>::~ShallowWaterEquationModel()
	{
	}

	template<typename TDataType>
	bool ShallowWaterEquationModel<TDataType>::initializeImpl()
	{
		int num = m_position.getElementCount();
		m_accel.setElementCount(num);
		h.setElementCount(num);
		h_buffer.setElementCount(num);
		xcount = num / zcount;
		grid_vel_x.setElementCount((xcount + 1) * (zcount + 2));
		grid_vel_z.setElementCount((xcount + 2) * (zcount + 1));
		grid_accel_x.setElementCount((xcount + 1) * (zcount + 2));
		grid_accel_z.setElementCount((xcount + 2) * (zcount + 1));

		printf("neighbor limit is 4, index count is %d\n", solid.getElementCount());
		cuint pDims = cudaGridSize(num, BLOCK_SIZE);
		cuint pDims2 = cudaGridSize((xcount + 2) * (zcount + 2), BLOCK_SIZE);
		Init <Real, Coord> << < pDims, BLOCK_SIZE >> > (m_position.getValue(), solid.getValue(), h.getValue(), isBound.getValue(), m_velocity.getValue());
		Init_gridVel <Real, Coord> << < pDims2, BLOCK_SIZE >> > (grid_vel_x.getValue(), grid_vel_z.getValue());
		cuSynchronize();
		return true;
	}

	__device__ int neighborFind(int ix, int iz, int j, int zcount) {
		if (j == 0) {
			if (iz == 0)return -1;
			else return ix * zcount + iz - 1;
		}
		else if (j == 1) {
			if (iz + 1 == zcount)return -1;
			else return ix * zcount + iz + 1;
		}
		else if (j == 2) {
			return (ix - 1)*zcount + iz;
		}
		else if (j == 3) {
			return (ix + 1)*zcount + iz;
		}
	}

	template<typename Real, typename Coord>
	__global__ void computeBoundConstrant(
		DeviceArray<Real> h,
		DeviceArray<int> xindex,
		DeviceArray<int> zindex,
		DeviceArray<Coord> m_accel,
		DeviceArray<Coord> m_velocity,
		DeviceArray<Coord> m_position,
		DeviceArray<int> isBound,
		int zcount,
		Real distance,
		Real gravity,
		Real dt)
	{
		int i = threadIdx.x + (blockIdx.x * blockDim.x);
		if (i >= h.size())  return;

		if (isBound[i] == 0)
			return;
		int maxNei = 4;

		int ix = xindex[i];
		int iz = zindex[i];

		for (int j = 0; j < maxNei; ++j)
		{
			int nei = neighborFind(ix, iz, j, zcount);
			if (nei >= h.size() || nei < 0)
			{
				switch (j)
				{
				case 0:
				case 1:
					m_velocity[i][2] = 0;
					break;
				case 2:
				case 3:
					m_velocity[i][0] = 0;
					break;
				}
			}

		}
	}

	template<typename Real, typename Coord>
	__global__ void computeAccel(
		DeviceArray<Real> h,
		DeviceArray<int> xindex,
		DeviceArray<int> zindex,
		DeviceArray<Coord> m_accel,
		DeviceArray<Coord> m_velocity,
		DeviceArray<Coord> m_position,
		DeviceArray<Coord> solid,
		DeviceArray<int> isBound,
		int zcount,
		Real distance,
		Real gravity,
		Real dt)
	{
		int i = threadIdx.x + (blockIdx.x * blockDim.x);
		if (i >= h.size())  return;
		int maxNei = 4;
		
		int ix = xindex[i];
		int iz = zindex[i];
		Real hx = 0, hz = 0;
		Real p1, p2;
		Real ux = 0, uz = 0, wx = 0, wz = 0;
		for (int j = 0; j < maxNei; ++j)
		{
			int nei = neighborFind(ix, iz, j, zcount);
			if (nei >= h.size() || nei < 0)
			{
				continue;
			}
			p1 = m_position[nei][1]; p2 = m_position[i][1];
			if (solid[nei][1] >= m_position[i][1] && h[nei] == 0)
				p1 = p2 = 0;
			if (solid[i][1] >= m_position[nei][1] && h[i] == 0)
				p1 = p2 = 0;

			if(j < maxNei/2)//gradient along z
			{
				hz += (p1 - p2) / (m_position[nei][2] - m_position[i][2]);
				uz += (m_velocity[nei][0] - m_velocity[i][0]) / (m_position[nei][2] - m_position[i][2]);
				wz += (m_velocity[nei][2] - m_velocity[i][2]) / (m_position[nei][2] - m_position[i][2]);
			}
			else
			{
				hx += (p1 - p2) / (m_position[nei][0] - m_position[i][0]);
				ux += (m_velocity[nei][0] - m_velocity[i][0]) / (m_position[nei][0] - m_position[i][0]);
				wx += (m_velocity[nei][2] - m_velocity[i][2]) / (m_position[nei][0] - m_position[i][0]);
			}
		}
		m_accel[i][0] = -gravity * hx / 2;
		m_accel[i][2] = -gravity * hz / 2;

		//m_accel[i][0] = -(gravity * hx + m_velocity[i][0] * ux) / 2 - m_velocity[i][2] * uz / 2;
		//m_accel[i][2] = -(gravity * hz + m_velocity[i][2] * wz) / 2 - m_velocity[i][0] * wx / 2;

		//Real maxAccel = 2 * distance / (dt*dt);
		//Real accel = sqrt(pow(m_accel[i][0], 2) + pow(m_accel[i][2], 2));
		//if(accel > maxAccel)
		//{
		//	printf("%d exceed max accel \n", i);
		//	m_accel[i][0] *= maxAccel / accel;
		//	m_accel[i][2] *= maxAccel / accel;
		//}
	}

	template<typename Real, typename Coord>
	__global__ void computeGridAccel(
		DeviceArray<Real> grid_vel_x,
		DeviceArray<Real> grid_vel_z,
		DeviceArray<Real> grid_accel_x,
		DeviceArray<Real> grid_accel_z,
		DeviceArray<Real> h,
		DeviceArray<Coord> m_position,
		DeviceArray<Coord> solid,
		int zcount,
		Real gravity,
		Real distance
	)
	{
		int i = threadIdx.x + (blockIdx.x * blockDim.x);
		Real hx = 0, hz = 0;
		Real p1, p2;
		Real ux = 0, uz = 0, wx = 0, wz = 0;
		int ix, iz;
		int xcount = m_position.size() / zcount;
		if (i < grid_vel_x.size())
		{
			ix = i / (zcount + 2);
			iz = i % (zcount + 2);
			if (iz == 0 || iz == zcount + 1 || ix == 0 || ix == xcount)
				grid_accel_x[i] = 0;
			else 
			{
				//particles
				int nei1 = (ix - 1) * zcount + iz - 1, nei2 = ix * zcount + iz - 1;
				p1 = m_position[nei1][1]; p2 = m_position[nei2][1];
				if (solid[nei1][1] >= m_position[nei2][1] && h[nei1] == 0)
					p1 = p2 = 0;
				if (solid[nei2][1] >= m_position[nei1][1] && h[nei2] == 0)
					p1 = p2 = 0;
				hx = (p1 - p2) / (m_position[nei1][0] - m_position[nei2][0]);

				//grid
				nei1 = i - 1, nei2 = i + 1;
				uz = (grid_vel_x[nei2] - grid_vel_x[nei1]) / distance * 0.5;
				nei1 = i - zcount - 2; nei2 = i + zcount + 2;
				ux = (grid_vel_x[nei2] - grid_vel_x[nei1]) / distance * 0.5;
				//compute velocity z
				//Real u = grid_vel_x[i], w = 0;
				Real u = grid_vel_x[i] * 0.5 + 
					0.25 * (grid_vel_x[(ix-1)*(zcount+2)+iz] + grid_vel_x[(ix + 1) * (zcount + 2) + iz]),
					w = 0;
				w += grid_vel_z[ix * (zcount + 1) + iz - 1]; w += grid_vel_z[ix * (zcount + 1) + iz];
				w += grid_vel_z[(ix + 1) * (zcount + 1) + iz - 1]; w += grid_vel_z[(ix + 1) * (zcount + 1) + iz];
				w *= 0.25;

				grid_accel_x[i] = -(u * ux + w * uz + gravity * hx);
				//grid_accel_x[i] = -(gravity * hx);
			}
		}
		if (i < grid_vel_z.size())
		{
			ix = i / (zcount + 1);
			iz = i % (zcount + 1);
			if (ix == 0 || iz == 0 || ix == xcount + 1 || iz == zcount)
				grid_accel_z[i] = 0;
			else
			{
				//particles
				int nei1 = (ix - 1) * zcount + iz - 1, nei2 = (ix - 1) * zcount + iz;
				p1 = m_position[nei1][1]; p2 = m_position[nei2][1];
				if (solid[nei1][1] >= m_position[nei2][1] && h[nei1] == 0)
					p1 = p2 = 0;
				if (solid[nei2][1] >= m_position[nei1][1] && h[nei2] == 0)
					p1 = p2 = 0;
				hz = (p1 - p2) / (m_position[nei1][2] - m_position[nei2][2]);

				//grid
				nei1 = i - 1, nei2 = i + 1;
				wz = (grid_vel_z[nei2] - grid_vel_z[nei1]) / distance * 0.5;
				nei1 = i - zcount - 1; nei2 = i + zcount + 1;
				wx = (grid_vel_z[nei2] - grid_vel_z[nei1]) / distance * 0.5;
				//compute velocity z
				//Real u = 0, w = grid_vel_z[i];
				Real u = 0,
					w = 0.5 * grid_vel_z[i] + 0.25 * (grid_vel_z[i - 1] + grid_vel_z[i + 1]);
				u += grid_vel_x[(ix - 1) * (zcount + 2) + iz]; u += grid_vel_x[(ix - 1) * (zcount + 2) + iz + 1];
				u += grid_vel_x[ix * (zcount + 2) + iz]; u += grid_vel_x[ix * (zcount + 2) + iz + 1];
				u *= 0.25;

				grid_accel_z[i] = -(u * wx + w * wz + gravity * hz);
				//grid_accel_z[i] = -gravity * hz;
			}
		}
	}
	template<typename Real, typename Coord>
	__global__ void computeGridVelocity(
		DeviceArray<Real> grid_vel_x,
		DeviceArray<Real> grid_vel_z,
		DeviceArray<Real> grid_accel_x,
		DeviceArray<Real> grid_accel_z,
		int zcount,
		Real relax,
		Real gravity,
		Real distance,
		Real dt
	)
	{
		int i = threadIdx.x + (blockIdx.x * blockDim.x);
		int ix, iz;
		int xcount = grid_vel_x.size() / (zcount + 2) - 1;
		//if (i == 0)
		//	printf("xcount is %d, zcount is %d\n", xcount, zcount);
		if (i < grid_vel_x.size())
		{
			ix = i / (zcount + 2);
			iz = i % (zcount + 2);
			//if (iz == 0 || iz == zcount + 1 || ix == 0 || ix == xcount)
			//	grid_vel_x[i] = 0;
			grid_vel_x[i] = grid_vel_x[i] * relax + grid_accel_x[i] * dt;
		}
		if (i < grid_vel_z.size())
		{
			ix = i / (zcount + 1);
			iz = i % (zcount + 1);
			//if (iz == 0 || iz == zcount || ix == 0 || ix == xcount + 1)
			//	grid_vel_z[i] = 0;
			grid_vel_z[i] = grid_vel_z[i] * relax + grid_accel_z[i] * dt;
		}
		Real maxVel = 2*sqrt(distance * gravity), vel;
		//vel = sqrt(pow(m_velocity[i][0], 2) + pow(m_velocity[i][2], 2));
		vel = abs(grid_vel_x[i]);
		if (vel > maxVel)
		{
			grid_vel_x[i] *= maxVel / vel;
		}
		vel = abs(grid_vel_z[i]);
		if (vel > maxVel)
		{
			grid_vel_z[i] *= maxVel / vel;
		}
	}

	template<typename Real, typename Coord>
	__global__ void computeVelocity(
		DeviceArray<Real> grid_vel_x,
		DeviceArray<Real> grid_vel_z,
		DeviceArray<Real> h,
		DeviceArray<Coord> m_position,
		DeviceArray<Coord> m_velocity,
		int zcount,
		Real distance,
		Real gravity,
		Real dt)
	{
		int i = threadIdx.x + (blockIdx.x * blockDim.x);
		if (i >= h.size())  return;
		int xcount = h.size() / zcount;
		int ix = i / zcount;
		int iz = i % zcount;
		
		m_velocity[i][0] = 0.5 * (grid_vel_x[ix * (zcount + 2) + iz + 1] + grid_vel_x[(ix + 1) * (zcount + 2) + iz + 1]);
		m_velocity[i][2] = 0.5 * (grid_vel_z[(ix + 1) * (zcount + 1) + iz] + grid_vel_z[(ix + 1) * (zcount + 1) + iz + 1]);
		if (ix == 0 || ix == xcount - 1)
			m_velocity[i][0] = 0;
		if (iz == 0 || iz == zcount - 1)
			m_velocity[i][2] = 0;

	}

	template<typename Real, typename Coord>
	__global__ void computeHeight(
		DeviceArray<Real> h,
		DeviceArray<Real> h_buffer,
		DeviceArray<int> xindex,
		DeviceArray<int> zindex,
		DeviceArray<Coord> m_velocity,
		DeviceArray<Coord> m_accel,
		DeviceArray<int> isBound,
		DeviceArray<Coord> m_position,
		DeviceArray<Coord> solid,
		DeviceArray<Coord> normal,
		int zcount,
		Real distance,
		Real dt)
	{
		int i = threadIdx.x + (blockIdx.x * blockDim.x);
		if (i >= h.size())  return;
		int maxNei = 4;
		int ix = xindex[i];
		int iz = zindex[i];

		Real uhx = 0, whz = 0;
		for (int j = 0; j < maxNei; ++j)
		{
			int nei = neighborFind(ix, iz, j, zcount);
			//bound cell
			if (nei >= h.size() || nei < 0)
			{
				continue;
			}
			if (j < maxNei / 2)//gradient along z
			{
				whz += (h[nei] * m_velocity[nei][2] - h[i] * m_velocity[i][2]) / (m_position[nei][2] - m_position[i][2]);
			}
			else
			{
				uhx += (h[nei] * m_velocity[nei][0] - h[i] * m_velocity[i][0]) / (m_position[nei][0] - m_position[i][0]);
			}
		}
		h_buffer[i] = -(uhx / 2 + whz / 2)*dt;
	}

	template<typename Real, typename Coord>
	__global__ void applyHeight(
		DeviceArray<Real> h,
		DeviceArray<Real> h_buffer,
		DeviceArray<Coord> m_position,
		DeviceArray<Coord> solid,
		DeviceArray<Coord> m_velocity
	)
	{
		int i = threadIdx.x + (blockIdx.x * blockDim.x);
		if (i >= h.size())  return;
		h[i] += h_buffer[i];
		if (h[i] < 0)
		{
			h[i] = 0;
			//m_velocity[i][0] = m_velocity[i][2] = 0;
			m_velocity[i][1] = max(0.0, m_velocity[i][1]);
		}
		m_position[i][1] = solid[i][1] + h[i];
	}
	template<typename TDataType>
	void ShallowWaterEquationModel<TDataType>::step(Real dt)
	{
		Node* parent = getParent();
		if (parent == NULL)
		{
			Log::sendMessage(Log::Error, "Parent not set for ParticleSystem!");
			return;
		}

		int num = m_position.getElementCount();
		cuint pDims = cudaGridSize(num, BLOCK_SIZE);
		cuint pDims2 = cudaGridSize(max((zcount+1)*(xcount+2), (zcount+2)*(xcount+1)), BLOCK_SIZE);

		////��ʱ��ʼ
		//hipEvent_t start, stop;
		//hipEventCreate(&start);
		//hipEventCreate(&stop);
		//hipEventRecord(start);

		computeGridAccel <Real, Coord> << < pDims2, BLOCK_SIZE >> > (
			grid_vel_x.getValue(),
			grid_vel_z.getValue(),
			grid_accel_x.getValue(),
			grid_accel_z.getValue(),
			h.getValue(),
			m_position.getValue(),
			solid.getValue(),
			zcount,
			9.8,
			distance
			);
		cuSynchronize();
		hipDeviceSynchronize();

		computeGridVelocity <Real, Coord> << < pDims2, BLOCK_SIZE >> > (
			grid_vel_x.getValue(),
			grid_vel_z.getValue(),
			grid_accel_x.getValue(),
			grid_accel_z.getValue(),
			zcount,
			relax,
			9.8,
			distance,
			dt
			);
		cuSynchronize();
		hipDeviceSynchronize();

		computeVelocity <Real, Coord> << < pDims, BLOCK_SIZE >> > (
			grid_vel_x.getValue(),
			grid_vel_z.getValue(),
			h.getValue(),
			m_position.getValue(),
			m_velocity.getValue(),
			zcount,
			distance,
			9.8,
			dt
			);
		cuSynchronize();
		hipDeviceSynchronize();

		computeHeight <Real, Coord> << < pDims, BLOCK_SIZE >> > (
			h.getValue(),
			h_buffer.getValue(),
			xindex.getValue(),
			zindex.getValue(),
			m_velocity.getValue(),
			m_accel.getValue(),
			isBound.getValue(),
			m_position.getValue(),
			solid.getValue(),
			normal.getValue(),
			zcount,
			distance,
			dt
			);
		cuSynchronize();
		hipDeviceSynchronize();

		applyHeight <Real, Coord> << < pDims, BLOCK_SIZE >> > (
			h.getValue(),
			h_buffer.getValue(),
			m_position.getValue(),
			solid.getValue(),
			m_velocity.getValue()
			);
		cuSynchronize();
		hipDeviceSynchronize();

		//hipEventRecord(stop);

		//hipEventSynchronize(stop);
		//float milliseconds = 0;
		//hipEventElapsedTime(&milliseconds, start, stop);

		//sumtimes += milliseconds;
		//sumnum++;
		//printf("Time: %f \n", sumtimes / sumnum);
		/*
		//�������
		if (sumnum == 1000) {
			std::string s = "C:\\temp.txt";
			std::ofstream myout;
			myout.open(s, std::ios::out | std::ios::app);
			if (!myout) {
				std::ofstream newout(s);
				newout << sumtimes / sumnum << std::endl;
			}
			myout << sumtimes / sumnum << std::endl;
			myout.close();
		}
		*/
	/*	hipblasHandle_t handle;
		float sum;
		hipblasCreate(&handle);
		hipblasSasum(handle, solid.getElementCount(), h.getValue().getDataPtr(), 1, &sum);
		hipblasDestroy(handle);
		printf("total height is %f\n", sum);*/
	}
}
