#include "hip/hip_runtime.h"
#include "FastMultiphaseSPH.h"

#include "Framework/Topology/PointSet.h"
#include "Core/Utility.h"

#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>

namespace PhysIKA
{
	IMPLEMENT_CLASS_1(FastMultiphaseSPH, TDataType)

		template<typename TDataType>
	FastMultiphaseSPH<TDataType>::FastMultiphaseSPH(std::string name)
		: Node(name)
	{
		//		attachField(&m_velocity, MechanicalState::velocity(), "Storing the particle velocities!", false);
		//		attachField(&m_force, MechanicalState::force(), "Storing the force densities!", false);

		m_pSet = std::make_shared<PointSet<TDataType>>();
		this->setTopologyModule(m_pSet);

		m_msph = std::make_shared<msph::MultiphaseSPHSolver>();
		m_msph->init();
		
		prepareData();
		
		std::vector<Coord> buffer(num_o);
		m_pSet->setPoints(buffer);
		m_pSet->setNormals(buffer);
		m_phase_concentration.setElementCount(num_o);

		updateTopology();
		// 		m_pointsRender = std::make_shared<PointRenderModule>();
		// 		this->addVisualModule(m_pointsRender);
	}
	struct OpaquePred {
		__host__ __device__
		bool operator()(Vector4f v) {
			return v[3] != 0;
		}
	};
	template<typename TDataType>
	void FastMultiphaseSPH<TDataType>::prepareData()
	{
		// get all particles
		int num = m_msph->num_particles;
		if (num != m_pos.size()) {
			m_pos.resize(num);
			m_color.resize(num);
		}
		m_msph->prepareRenderData((cfloat3*)m_pos.getDataPtr(), (cfloat4*)m_color.getDataPtr());
		num_o = num;
		// then filter transparent particles
		Vector3f* d_pos = m_pos.getDataPtr();
		Vector4f* d_color = m_color.getDataPtr();
		thrust::copy_if(thrust::device, d_pos, d_pos + num, d_color, d_pos, OpaquePred());
		auto oe = thrust::copy_if(thrust::device, d_color, d_color + num, d_color, OpaquePred());
		num_o = oe - d_color;
	}

	template<typename TDataType>
	FastMultiphaseSPH<TDataType>::~FastMultiphaseSPH()
	{

	}

	template<typename TDataType>
	void FastMultiphaseSPH<TDataType>::advance(Real dt)
	{
		// dt not used here as its managed by external solver ...
		m_msph->step();
	}


	template<typename TDataType>
	void FastMultiphaseSPH<TDataType>::loadParticles(std::string filename)
	{
		m_pSet->loadObjFile(filename);
	}

	template<typename TDataType>
	void FastMultiphaseSPH<TDataType>::loadParticles(Coord center, Real r, Real distance)
	{
		std::vector<Coord> vertList;
		std::vector<Coord> normalList;

		Coord lo = center - r;
		Coord hi = center + r;

		for (Real x = lo[0]; x <= hi[0]; x += distance)
		{
			for (Real y = lo[1]; y <= hi[1]; y += distance)
			{
				for (Real z = lo[2]; z <= hi[2]; z += distance)
				{
					Coord p = Coord(x, y, z);
					if ((p - center).norm() < r)
					{
						vertList.push_back(Coord(x, y, z));
					}
				}
			}
		}
		normalList.resize(vertList.size());

		m_pSet->setPoints(vertList);
		m_pSet->setNormals(normalList);

		vertList.clear();
		normalList.clear();
	}

	template<typename TDataType>
	void FastMultiphaseSPH<TDataType>::loadParticles(Coord lo, Coord hi, Real distance)
	{
		std::vector<Coord> vertList;
		std::vector<Coord> normalList;

		for (Real x = lo[0]; x <= hi[0]; x += distance)
		{
			for (Real y = lo[1]; y <= hi[1]; y += distance)
			{
				for (Real z = lo[2]; z <= hi[2]; z += distance)
				{
					Coord p = Coord(x, y, z);
					vertList.push_back(Coord(x, y, z));
				}
			}
		}
		normalList.resize(vertList.size());

		m_pSet->setPoints(vertList);
		m_pSet->setNormals(normalList);

		std::cout << "particle number: " << vertList.size() << std::endl;

		vertList.clear();
		normalList.clear();
	}

	template<typename TDataType>
	bool FastMultiphaseSPH<TDataType>::translate(Coord t)
	{
		m_pSet->translate(t);

		return true;
	}


	template<typename TDataType>
	bool FastMultiphaseSPH<TDataType>::scale(Real s)
	{
		m_pSet->scale(s);

		return true;
	}

	template<typename TDataType>
	bool FastMultiphaseSPH<TDataType>::initialize()
	{
		return Node::initialize();
	}

	// 	template<typename TDataType>
	// 	void FastMultiphaseSPH<TDataType>::setVisible(bool visible)
	// 	{
	// 		if (m_pointsRender == nullptr)
	// 		{
	// 			m_pointsRender = std::make_shared<PointRenderModule>();
	// 			this->addVisualModule(m_pointsRender);
	// 		}
	// 
	// 		Node::setVisible(visible);
	// 	}

	struct ColorOp {
		__host__ __device__
			Vector3f operator()(Vector4f color) {
			return Vector3f(1 - color[0], 1 - color[0], 1 - color[0]);
		}
	};

	template<typename TDataType>
	void FastMultiphaseSPH<TDataType>::updateTopology()
	{
		//if (!this->currentPosition()->isEmpty())
		//{
		//	int num = this->currentPosition()->getElementCount();
		//	auto& pts = m_pSet->getPoints();
		//	if (num != pts.size())
		//	{
		//		pts.resize(num);
		//	}
		//	Function1Pt::copy(pts, this->currentPosition()->getValue());
		//}
		prepareData();
		auto pts = m_pSet->getPoints();
		hipMemcpy(pts.getDataPtr(), m_pos.getDataPtr(), sizeof(Coord) * num_o, hipMemcpyDeviceToDevice);
		Vector3f* color_idx = m_phase_concentration.getValue().getDataPtr();
		thrust::transform(thrust::device, m_color.getDataPtr(), m_color.getDataPtr() + num_o, color_idx, ColorOp());
	}


	template<typename TDataType>
	bool FastMultiphaseSPH<TDataType>::resetStatus()
	{
		auto pts = m_pSet->getPoints();

		if (pts.size() > 0)
		{
			this->currentPosition()->setElementCount(pts.size());
			this->currentVelocity()->setElementCount(pts.size());
			this->currentForce()->setElementCount(pts.size());

			Function1Pt::copy(this->currentPosition()->getValue(), pts);
			this->currentVelocity()->getReference()->reset();
		}

		return Node::resetStatus();
	}

	// 	template<typename TDataType>
	// 	std::shared_ptr<PointRenderModule> FastMultiphaseSPH<TDataType>::getRenderModule()
	// 	{
	// // 		if (m_pointsRender == nullptr)
	// // 		{
	// // 			m_pointsRender = std::make_shared<PointRenderModule>();
	// // 			this->addVisualModule(m_pointsRender);
	// // 		}
	// 
	// 		return m_pointsRender;
	// 	}
}